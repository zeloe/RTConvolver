#include "hip/hip_runtime.h"

#include "GPUConvEngine256.cuh"
// Define the constant memory array
__constant__ int SIZES_256[3];
 
__constant__ float INPUT_256[256];
__constant__ float INPUT2_256[256]; 

__shared__ float partArray_256_1[256 * 4];
__shared__ float partArray_256_2[256 * 4];
__shared__ float partArray_256_3[256 * 4];
__shared__ float partArray_256_4[256 * 4];
__constant__ int OFFSETS_256[3];

__global__ void  shared_partitioned_convolution_256_4(float* __restrict__ Result, const float* __restrict__ Dry, const float* __restrict__ Imp) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int partition_idx = blockIdx.x;
	const unsigned int copy_idx = threadIdx.x;


	// Declare pointers to the shared memory partitions
	float* arr1 = &partArray_256_4[0];
	float* arr2 = &partArray_256_4[SIZES_256[0]];
	float* tempResult = &partArray_256_4[SIZES_256[0] * 2];
	// Load data into shared memory
	tempResult[copy_idx] = 0.f;
	tempResult[SIZES_256[0] + copy_idx] = 0.f;


	arr1[copy_idx] = Dry[thread_idx + OFFSETS_256[2]];
	arr2[copy_idx] = Imp[thread_idx + OFFSETS_256[2]];

	// Shared memory to accumulate results before writing them to global memory
	// Convolution operation (reduction into shared memory)
#pragma unroll
	for (int i = 0; i < SIZES_256[0]; i++) {
		int inv = (i - copy_idx) % SIZES_256[0];
		tempResult[i + inv] += arr1[i] * arr2[inv];
	}

	__syncthreads();  // Ensure all threads in the block have finished processing


	// Write the accumulated result to global memory (only for the first thread)
	if (copy_idx == 0) {
		// Write the first part of the result (up to SIZES[0] * 2 - 1)
#pragma unroll
		for (int i = 0; i < SIZES_256[1]; i++) {
			atomicAdd(&Result[i], tempResult[i]);
		}
	}

}





__global__ void  shared_partitioned_convolution_256_3(float* __restrict__ Result, const float* __restrict__ Dry, const float* __restrict__ Imp) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int partition_idx = blockIdx.x;
	const unsigned int copy_idx = threadIdx.x;


	// Declare pointers to the shared memory partitions
	float* arr1 = &partArray_256_3[0];
	float* arr2 = &partArray_256_3[SIZES_256[0]];
	float* tempResult = &partArray_256_3[SIZES_256[0] * 2];
	// Load data into shared memory
	tempResult[copy_idx] = 0.f;
	tempResult[SIZES_256[0] + copy_idx] = 0.f;


	arr1[copy_idx] = Dry[thread_idx + OFFSETS_256[1]];
	arr2[copy_idx] = Imp[thread_idx + OFFSETS_256[1]];

	// Shared memory to accumulate results before writing them to global memory
	// Convolution operation (reduction into shared memory)
#pragma unroll
	for (int i = 0; i < SIZES_256[0]; i++) {
		int inv = (i - copy_idx) % SIZES_256[0];
		tempResult[i + inv] += arr1[i] * arr2[inv];
	}

	__syncthreads();  // Ensure all threads in the block have finished processing


	// Write the accumulated result to global memory (only for the first thread)
	if (copy_idx == 0) {
		// Write the first part of the result (up to SIZES[0] * 2 - 1)
#pragma unroll
		for (int i = 0; i < SIZES_256[1]; i++) {
			atomicAdd(&Result[i], tempResult[i]);
		}
	}

}



__global__ void  shared_partitioned_convolution_256_2(float* __restrict__ Result, const float* __restrict__ Dry, const float* __restrict__ Imp) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int partition_idx = blockIdx.x;
	const unsigned int copy_idx = threadIdx.x;


	// Declare pointers to the shared memory partitions
	float* arr1 = &partArray_256_2[0];
	float* arr2 = &partArray_256_2[SIZES_256[0]];
	float* tempResult = &partArray_256_2[SIZES_256[0] * 2];
	// Load data into shared memory
	tempResult[copy_idx] = 0.f;
	tempResult[SIZES_256[0] + copy_idx] = 0.f;


	arr1[copy_idx] = Dry[thread_idx + OFFSETS_256[0]];
	arr2[copy_idx] = Imp[thread_idx + OFFSETS_256[0]];

	// Shared memory to accumulate results before writing them to global memory
	// Convolution operation (reduction into shared memory)
#pragma unroll
	for (int i = 0; i < SIZES_256[0]; i++) {
		int inv = (i - copy_idx) % SIZES_256[0];
		tempResult[i + inv] += arr1[i] * arr2[inv];
	}

	__syncthreads();  // Ensure all threads in the block have finished processing


	// Write the accumulated result to global memory (only for the first thread)
	if (copy_idx == 0) {
		// Write the first part of the result (up to SIZES[0] * 2 - 1)
#pragma unroll
		for (int i = 0; i < SIZES_256[1]; i++) {
			atomicAdd(&Result[i], tempResult[i]);
		}
	}

}




__global__ void  shared_partitioned_convolution_256(float* __restrict__ Result, const float* __restrict__ Dry, const float* __restrict__ Imp) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int partition_idx = blockIdx.x;
	const unsigned int copy_idx = threadIdx.x;
	 

	// Declare pointers to the shared memory partitions
	float* arr1 = &partArray_256_1[0];
	float* arr2 = &partArray_256_1[SIZES_256[0]];
	float* tempResult = &partArray_256_1[SIZES_256[0] * 2];
	// Load data into shared memory
	tempResult[copy_idx] = 0.f;
	tempResult[SIZES_256[0] + copy_idx] = 0.f;


	arr1[copy_idx] = Dry[thread_idx];
	arr2[copy_idx] = Imp[thread_idx];

	// Shared memory to accumulate results before writing them to global memory
	// Convolution operation (reduction into shared memory)
#pragma unroll
	for (int i = 0; i < SIZES_256[0]; i++) {
		int inv = (i - copy_idx) % SIZES_256[0];
		tempResult[i + inv] += arr1[i] * arr2[inv];
	}

	__syncthreads();  // Ensure all threads in the block have finished processing


	// Write the accumulated result to global memory (only for the first thread)
	if (copy_idx == 0) {
		// Write the first part of the result (up to SIZES[0] * 2 - 1)
#pragma unroll
		for (int i = 0; i < SIZES_256[1]; i++) {
			atomicAdd(&Result[i], tempResult[i]);
		}
	}

}

__global__ void  shiftAndInsertKernel_256(float* __restrict__ delayBuffer) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	// Insert new elements at the beginning of the delay buffer
	if (tid < SIZES_256[0]) {
		delayBuffer[tid] = INPUT_256[tid];
	}
	
		delayBuffer[tid + SIZES_256[0]] = delayBuffer[tid];
}


__global__ void  shiftAndInsertKernel2_256(float* __restrict__ delayBuffer) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	// Insert new elements at the beginning of the delay buffer
	if (tid < SIZES_256[0]) {
		delayBuffer[tid] = INPUT2_256[tid];
	}

	delayBuffer[tid + SIZES_256[0]] = delayBuffer[tid];


} 



GPUConvEngine_256::GPUConvEngine_256() {
	hipStreamCreate(&stream);
	dThreads.x = maxBufferSize;
	sizeMax = (((48000 * 6) / maxBufferSize) + 1) * maxBufferSize;
	h_convResSize = maxBufferSize * 2;
	floatSizeRes = h_convResSize * sizeof(float);
	(hipMalloc((void**)&d_ConvolutionResL, floatSizeRes));
	(hipMalloc((void**)&d_ConvolutionResR, floatSizeRes));
	h_numPartitions = sizeMax / maxBufferSize;
	 
 
	bs_float = maxBufferSize * sizeof(float);

	


	h_ConvolutionResL = (float*)calloc(h_convResSize, sizeof(float));
	h_ConvolutionResR = (float*)calloc(h_convResSize, sizeof(float));
	h_OverlapL = (float*)calloc(maxBufferSize, sizeof(float));
	h_OverlapR = (float*)calloc(maxBufferSize, sizeof(float));
	h_index = 0;

	cpu_sizes = (int*)calloc(3, sizeof(int));
	cpu_offsets = (int*)calloc(3, sizeof(int));
	cpu_sizes[0] = maxBufferSize;
	cpu_sizes[1] = h_convResSize;
	cpu_sizes[2] = h_numPartitions;
	hipMemcpyToSymbol(HIP_SYMBOL(SIZES_256), cpu_sizes, 3 * sizeof(int));


	 
	h_paddedSize = h_numPartitions * maxBufferSize;
	  
	cpu_sizes[1] = h_convResSize;
	hipMemcpyToSymbol(HIP_SYMBOL(SIZES_256), cpu_sizes, 3 * sizeof(int));  


	(hipMalloc((void**)&d_IR_paddedL, h_paddedSize * sizeof(float)));
	(hipMalloc((void**)&d_IR_paddedR, h_paddedSize * sizeof(float)));

	(hipMalloc((void**)&d_TimeDomain_paddedL, h_paddedSize * sizeof(float)));
	(hipMalloc((void**)&d_TimeDomain_paddedR, h_paddedSize * sizeof(float)));
	
	clear();
	
}
void GPUConvEngine_256::clear() {
	int floatSizeResMax = maxBufferSize * sizeof(float);
	(hipMemset(d_ConvolutionResL, 0, floatSizeResMax));
	(hipMemset(d_ConvolutionResR, 0, floatSizeResMax));
	(hipMemset(INPUT_256, 0, maxBufferSize * sizeof(float)));
	(hipMemset(INPUT2_256, 0, maxBufferSize * sizeof(float)));
	(hipMemset(d_IR_paddedL, 0, h_paddedSize * sizeof(float)));
	(hipMemset(d_IR_paddedR, 0, h_paddedSize * sizeof(float)));
	(hipMemset(d_TimeDomain_paddedL, 0, h_paddedSize * sizeof(float)));
	(hipMemset(d_TimeDomain_paddedR, 0, h_paddedSize * sizeof(float)));

}


GPUConvEngine_256::~GPUConvEngine_256() {
	cleanup();
	// Free Stream 
	hipStreamDestroy(stream);
}

void GPUConvEngine_256::cleanup() {
	
	hipFree(d_ConvolutionResL);
	hipFree(d_ConvolutionResR);
	 
	hipFree(d_IR_paddedL);
	hipFree(d_IR_paddedR);
	hipFree(d_TimeDomain_paddedL);
	hipFree(d_TimeDomain_paddedR);

	// Free CPU memory
	free(h_ConvolutionResL);
	free(h_ConvolutionResR);
	free(h_OverlapL);
	free(h_OverlapR);
	free(cpu_sizes);
	free(cpu_offsets);

 



}


void GPUConvEngine_256::checkCudaError(hipError_t err, const char* errMsg) {
	if (err != hipSuccess) {
		printf("CUDA Error (%s): %s\n", errMsg, hipGetErrorString(err));
	}
}

 

void GPUConvEngine_256::prepare(float size) {


	hipStreamSynchronize(stream);
	// Ensure proper padding
	int temp_h_paddedSize = (((size) / maxBufferSize) / numKernels + 1) * maxBufferSize * numKernels;
	int temp_numPartitions = temp_h_paddedSize / maxBufferSize / numKernels;
	int offset = temp_h_paddedSize / numKernels;
	cpu_offsets[0] = offset;
	cpu_offsets[1] = offset * 2;
	cpu_offsets[2] = offset * 3;
	hipMemcpyToSymbol(HIP_SYMBOL(OFFSETS_256), cpu_offsets, sizeof(int) * 3);
	convBlocks.x = temp_numPartitions;

  	h_numPartitions = (((size) / maxBufferSize) + 1);
	// Update dBlocks and other parameters
	dBlocks.x = h_numPartitions;

	// Update sizes array
	cpu_sizes[2] = h_numPartitions;

	// Copy updated sizes to device
	hipMemcpyToSymbol(HIP_SYMBOL(SIZES_256), cpu_sizes, sizeof(int) * 3);

}


void  GPUConvEngine_256::process(const float* in, const float* in2, const float* in3, const float* in4, float* out1, float* out2)  {
	 
	//copy content and transfer
	int indexBs = h_index * maxBufferSize;
	hipMemcpyToSymbolAsync(HIP_SYMBOL(INPUT_256), in, bs_float,0, hipMemcpyHostToDevice,stream);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(INPUT2_256), in2, bs_float,0, hipMemcpyHostToDevice,stream);
	hipMemcpyAsync(d_IR_paddedL + indexBs, in3, bs_float, hipMemcpyHostToDevice , stream);
	hipMemcpyAsync(d_IR_paddedR + indexBs, in4, bs_float, hipMemcpyHostToDevice, stream);



	//launch the convolution Engine
	launchEngine();

	   

	for (int i = 0; i < maxBufferSize; i += 4) {
		// Load 4 floats from h_ConvolutionResL and h_OverlapL
		__m128 resL = _mm_loadu_ps(&h_ConvolutionResL[i]);
		__m128 overlapL = _mm_loadu_ps(&h_OverlapL[i]);

		// Perform (resL + overlapL) 
		__m128 resultL = _mm_add_ps(resL, overlapL);
		_mm_storeu_ps(&out1[i], resultL); // Store the result in out1

		// Load 4 floats from h_ConvolutionResR and h_OverlapR
		__m128 resR = _mm_loadu_ps(&h_ConvolutionResR[i]);
		__m128 overlapR = _mm_loadu_ps(&h_OverlapR[i]);

		// Perform (resR + overlapR)
		__m128 resultR = _mm_add_ps(resR, overlapR);
		_mm_storeu_ps(&out2[i], resultR); // Store the result in out2

	 
	}
	
	// Copy the last `bs` elements as overlap values for the next block
	std::memcpy(h_OverlapL, &h_ConvolutionResL[maxBufferSize -  1 ], bs_float);
 	std::memcpy(h_OverlapR, &h_ConvolutionResR[maxBufferSize -  1 ], bs_float);

}




void  GPUConvEngine_256::launchEngine() {

	shiftAndInsertKernel_256 << <dBlocks, dThreads,0,stream >> > (d_TimeDomain_paddedL);
	shiftAndInsertKernel2_256 << <dBlocks, dThreads, 0, stream >> > (d_TimeDomain_paddedR);
	shared_partitioned_convolution_256 << <convBlocks,dThreads , 0, stream >> > (d_ConvolutionResL, d_TimeDomain_paddedL, d_IR_paddedL);
	shared_partitioned_convolution_256 << <convBlocks, dThreads, 0, stream >> > (d_ConvolutionResR, d_TimeDomain_paddedR, d_IR_paddedR);
	shared_partitioned_convolution_256_2 << <convBlocks, dThreads, 0, stream >> > (d_ConvolutionResL, d_TimeDomain_paddedL, d_IR_paddedL);
	shared_partitioned_convolution_256_2 << <convBlocks, dThreads, 0, stream >> > (d_ConvolutionResR, d_TimeDomain_paddedR, d_IR_paddedR);
	shared_partitioned_convolution_256_3 << <convBlocks, dThreads, 0, stream >> > (d_ConvolutionResL, d_TimeDomain_paddedL, d_IR_paddedL);
	shared_partitioned_convolution_256_3 << <convBlocks, dThreads, 0, stream >> > (d_ConvolutionResR, d_TimeDomain_paddedR, d_IR_paddedR);
	shared_partitioned_convolution_256_4 << <convBlocks, dThreads, 0, stream >> > (d_ConvolutionResL, d_TimeDomain_paddedL, d_IR_paddedL);
	shared_partitioned_convolution_256_4 << <convBlocks, dThreads, 0, stream >> > (d_ConvolutionResR, d_TimeDomain_paddedR, d_IR_paddedR);
	hipMemcpyAsync(h_ConvolutionResL, d_ConvolutionResL, floatSizeRes, hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(h_ConvolutionResR, d_ConvolutionResR, floatSizeRes, hipMemcpyDeviceToHost, stream);
	hipMemsetAsync(d_ConvolutionResL, 0, floatSizeRes, stream);
	hipMemsetAsync(d_ConvolutionResR, 0, floatSizeRes, stream);

	hipStreamSynchronize(stream);
	//update index

	h_index = (h_index + 1) % (h_numPartitions);
	
}
