#include "hip/hip_runtime.h"

#include "GPUConvEngine512.cuh" 

__constant__ int SIZES_512[3];
__constant__ float INPUT_512[512];
__constant__ float INPUT2_512[512];

__global__ void shared_partitioned_convolution_512(float* __restrict__ Result, const float* __restrict__ Dry, const float* __restrict__ Imp) {
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int copy_idx = threadIdx.x;
	extern __shared__ float partArray[];

	// Declare pointers to the shared memory partitions
	float* arr1 = &partArray[0];
	float* arr2 = &partArray[SIZES_512[0]];
	float* tempResult = &partArray[SIZES_512[0] * 2];
	// Load data into shared memory
	tempResult[copy_idx] = 0.f;
	tempResult[copy_idx + SIZES_512[0]] = 0.f;
	arr1[copy_idx] = Dry[thread_idx];
	arr2[copy_idx] = Imp[thread_idx];

	 
	__syncthreads();  // Ensure all threads in the block have finished processing
	#pragma unroll
	for (int i = 0; i < SIZES_512[0]; i++) {
		int inv = (i + SIZES_512[0] - copy_idx) % SIZES_512[0];
		tempResult[i + inv] += arr1[i] * arr2[inv];
	}
	__syncthreads();  // Ensure all threads in the block have finished processing

	// Write the accumulated result to global memory (only for the first thread)
	if (copy_idx == 0) {
		// Write the result into global memory
		#pragma unroll
		for (int i = 0; i < SIZES_512[1]; i++) {
			atomicAdd(&Result[i], tempResult[i]);
		}


	}

}

__global__ void  shiftAndInsertKernel_512(float* __restrict__ delayBuffer) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	// Insert new elements at the beginning of the delay buffer
	if (tid < SIZES_512[0]) {
		delayBuffer[tid] = INPUT_512[tid];
	}

	delayBuffer[tid + SIZES_512[0]] = delayBuffer[tid];
}


__global__ void  shiftAndInsertKernel2_512(float* __restrict__ delayBuffer) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	// Insert new elements at the beginning of the delay buffer
	if (tid < SIZES_512[0]) {
		delayBuffer[tid] = INPUT2_512[tid];
	}

	delayBuffer[tid + SIZES_512[0]] = delayBuffer[tid];


}


GPUConvEngine_512::GPUConvEngine_512() {
	hipStreamCreate(&stream);
	bs = maxBufferSize;
	sizeMax = (((96000) / maxNumberOfTheads) + 1) * maxNumberOfTheads;
	h_convResSize = bs * 2 - 1;
	floatSizeRes = h_convResSize * sizeof(float);
	(hipMalloc((void**)&d_ConvolutionResL, floatSizeRes));
	(hipMalloc((void**)&d_ConvolutionResR, floatSizeRes));

	SHMEM = 4 * maxNumberOfTheads * sizeof(float);
 
	bs_float = bs * sizeof(float);
	cpu_sizes = (int*)calloc(2, sizeof(int));

	h_ConvolutionResL = (float*)calloc(h_convResSize, sizeof(float));
	h_ConvolutionResR = (float*)calloc(h_convResSize, sizeof(float));
	h_OverlapL = (float*)calloc(bs, sizeof(float));
	h_OverlapR = (float*)calloc(bs, sizeof(float));
	h_index = 0; 
	cpu_sizes[0] = bs;
	cpu_sizes[1] = h_convResSize;


	h_numPartitions = sizeMax / bs;
	h_paddedSize = h_numPartitions * bs;

	 
	hipMemcpyToSymbol(HIP_SYMBOL(SIZES_512), cpu_sizes, 2 * sizeof(int));


	(hipMalloc((void**)&d_IR_paddedL, h_paddedSize * sizeof(float)));
	(hipMalloc((void**)&d_IR_paddedR, h_paddedSize * sizeof(float)));

	(hipMalloc((void**)&d_TimeDomain_paddedL, h_paddedSize * sizeof(float)));
	(hipMalloc((void**)&d_TimeDomain_paddedR, h_paddedSize * sizeof(float)));
	dThreads.x = maxNumberOfTheads;
	bs_float = bs * sizeof(float);
	h_convResSize = bs * 2 - 1;
	floatSizeRes = h_convResSize * sizeof(float);
	clear();
	free(cpu_sizes);
	 
}
void GPUConvEngine_512::clear() {
	int floatSizeResMax = maxBufferSize * sizeof(float);
	(hipMemset(d_ConvolutionResL, 0, floatSizeResMax));
	(hipMemset(d_ConvolutionResR, 0, floatSizeResMax));
	(hipMemset(INPUT_512, 0, maxBufferSize * sizeof(float)));
	(hipMemset(INPUT2_512, 0, maxBufferSize * sizeof(float)));
	(hipMemset(d_IR_paddedL, 0, h_paddedSize * sizeof(float)));
	(hipMemset(d_IR_paddedR, 0, h_paddedSize * sizeof(float)));
	(hipMemset(d_TimeDomain_paddedL, 0, h_paddedSize * sizeof(float)));
	(hipMemset(d_TimeDomain_paddedR, 0, h_paddedSize * sizeof(float)));

}


GPUConvEngine_512::~GPUConvEngine_512() {
	 
	cleanup();
	// Free Stream 
	hipStreamDestroy(stream);
}

void GPUConvEngine_512::cleanup() {
	
	hipFree(d_ConvolutionResL);
	hipFree(d_ConvolutionResR);
	 
	hipFree(d_IR_paddedL);
	hipFree(d_IR_paddedR);
	hipFree(d_TimeDomain_paddedL);
	hipFree(d_TimeDomain_paddedR);

	// Free CPU memory
	free(h_ConvolutionResL);
	free(h_ConvolutionResR);
	free(h_OverlapL);
	free(h_OverlapR);

 



}


void GPUConvEngine_512::checkCudaError(hipError_t err, const char* errMsg) {
	if (err != hipSuccess) {
		printf("CUDA Error (%s): %s\n", errMsg, hipGetErrorString(err));
	}
}

void GPUConvEngine_512::prepare(int sampleRate) {

	
	
	h_paddedSize = ((sampleRate / maxNumberOfTheads) + 1) * maxNumberOfTheads;
	h_numPartitions = h_paddedSize / maxNumberOfTheads;
	
	dBlocks.x = (h_numPartitions);

	threadsPerBlock.x = bs;
	numBlocks.x = (h_paddedSize + threadsPerBlock.x - 1) / threadsPerBlock.x;

	threadsPerBlockZero = bs;
	clear();
	
}

 



void  GPUConvEngine_512::process(const float* in, const float* in2, const float* in3, const float* in4, float* out1, float* out2) {
 
	//copy content and transfer
	int indexBs = h_index * bs;
	hipMemcpyToSymbolAsync(HIP_SYMBOL(INPUT_512), in, bs_float,0, hipMemcpyHostToDevice,stream);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(INPUT2_512), in2, bs_float,0, hipMemcpyHostToDevice,stream);
	hipMemcpyAsync(d_IR_paddedL + indexBs, in3, bs_float, hipMemcpyHostToDevice , stream);
	hipMemcpyAsync(d_IR_paddedR + indexBs, in4, bs_float, hipMemcpyHostToDevice, stream);
	


	//launch the convolution Engine
	launchEngine();

	for (int i = 0; i < bs; i += 4) {
		
	 
		// Load 4 floats from h_ConvolutionResL and h_OverlapL
		__m128 resL = _mm_loadu_ps(&h_ConvolutionResL[i]);
		 
		 
		__m128 overlapL = _mm_loadu_ps(&h_OverlapL[i]);

		// Perform (resL + overlapL) 
		__m128 resultL = _mm_add_ps(resL, overlapL);
		_mm_storeu_ps(&out1[i], resultL); // Store the result in out1

		// Load 4 floats from h_ConvolutionResR and h_OverlapR
		__m128 resR = _mm_loadu_ps(&h_ConvolutionResR[i]);
		__m128 overlapR = _mm_loadu_ps(&h_OverlapR[i]);
	 
		 
		// Perform (resR + overlapR)
		__m128 resultR = _mm_add_ps(resR, overlapR);
		_mm_storeu_ps(&out2[i], resultR); // Store the result in out2

	 
	}
	
	// Copy the last `bs` elements as overlap values for the next block
	std::memcpy(h_OverlapL, &h_ConvolutionResL[bs -  1 ], bs_float);
 	std::memcpy(h_OverlapR, &h_ConvolutionResR[bs -  1 ], bs_float);
	 
}




void  GPUConvEngine_512::launchEngine() {
	
	shiftAndInsertKernel_512 << <numBlocks, threadsPerBlock,0,stream >> > (d_TimeDomain_paddedL);
	shiftAndInsertKernel2_512 << <numBlocks, threadsPerBlock, 0, stream >> > (d_TimeDomain_paddedR);
	shared_partitioned_convolution_512 << <dBlocks,dThreads , SHMEM, stream >> > (d_ConvolutionResL, d_TimeDomain_paddedL, d_IR_paddedL);
	shared_partitioned_convolution_512 << <dBlocks, dThreads, SHMEM, stream >> > (d_ConvolutionResR, d_TimeDomain_paddedR, d_IR_paddedR);
	hipMemcpyAsync(h_ConvolutionResL, d_ConvolutionResL, floatSizeRes, hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(h_ConvolutionResR, d_ConvolutionResR, floatSizeRes, hipMemcpyDeviceToHost, stream);
	hipMemsetAsync(d_ConvolutionResL, 0, floatSizeRes, stream);
	hipMemsetAsync(d_ConvolutionResR, 0, floatSizeRes, stream);

	hipStreamSynchronize(stream);
	//update index

	h_index = (h_index + 1) % (h_numPartitions);
	
}
